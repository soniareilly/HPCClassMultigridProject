#include <stdio.h>
#include "gscu.h"
//#include "gs.h"


// Restriction test
int main()
{
    int N = 8;
    int n = N+1;
    int n2 = N/2+1;
    int i,j;
    double *arr = (double*) malloc ( sizeof(double) * n * n );
    double *sol = (double*) malloc ( sizeof(double) * n2 * n2 );
    for (i = 0; i < n*n; ++i) arr[i] = i;
    double *cuarr, *cusol;
    hipMalloc(&cuarr, sizeof(double) * n*n);
    hipMalloc(&cusol, sizeof(double) * n2*n2);
    hipMemcpy(cuarr, arr, n*n*sizeof(double), hipMemcpyHostToDevice);

    dim3 numthreads(n2,n2);
    restriction<<<numthreads,1>>>(cusol,cuarr,n);
    hipMemcpy(sol, cusol, n2*n2*sizeof(double), hipMemcpyDeviceToHost);

    for (i = 0; i < n2; +i)
    {
        for (j = 0; j < n2; ++j)
        {
            printf("%g\t",sol[i*n2+j]);
        }
        printf("\n");
    }

    free(arr);
    free(sol);
    hipFree(cuarr);
    hipFree(cusol);
}